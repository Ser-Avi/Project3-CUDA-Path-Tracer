#include "hip/hip_runtime.h"
#include "utils.h"

//__global__ void kernComputeIndices(int N, int* indices, int* gridIndices) {
//    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
//    if (index < N)
//    {
//        gridIndices[index] = ;// gridIndex3Dto1D(relativePos.x, relativePos.y, relativePos.z, gridResolution);
//    }
//}

namespace Utils
{
    __global__ void kernResetIntBuffer(int N, int* intBuffer, int value) {
        int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (index < N) {
            intBuffer[index] = value;
        }
    }

    __global__ void kernIdentifyStartEnd(int N, ShadeableIntersection* intSects,
        int* materialStartIndices, int* materialEndIndices) {
        int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (index > N - 1) return;

        MaterialType thisMat = intSects[index].materialType;

        if (index > 0 && index < N - 1 && thisMat != intSects[index - 1].materialType)
        {
            materialStartIndices[thisMat] = index;
            materialEndIndices[thisMat + 1] = index - 1;
        }
        else if (index == 0)
        {
            materialStartIndices[thisMat] = index;
        }
        else if (index == N - 1)
        {
            materialEndIndices[thisMat] = index;
        }
    }
}





//namespace StreamCompaction {
//
//    /**
//        * Maps an array to an array of 0s and 1s for stream compaction. Elements
//        * which map to 0 will be removed, and elements which map to 1 will be kept.
//        */
//    __global__ void kernMapToBoolean(int n, int* bools, const PathSegment* idata, std::function<bool(const PathSegment&)> predicate) {
//        int idx = blockIdx.x * blockDim.x + threadIdx.x;
//        if (idx > n - 1 || idx < 0) return;
//        bools[idx] = predicate(idata[idx]);
//    }
//
//    /**
//     * Performs scatter on an array. That is, for each element in idata,
//     * if bools[idx] == 1, it copies idata[idx] to odata[indices[idx]].
//     */
//    __global__ void kernScatter(int n, PathSegment* odata,
//        const PathSegment* idata, const int* bools, const int* indices) {
//        // TODO
//        int idx = blockIdx.x * blockDim.x + threadIdx.x;
//        if (idx > n - 1 || idx < 0) return;
//        if (bools[idx] == 1)
//        {
//            odata[indices[idx]] = idata[idx];
//        }
//    }
//
//    /// <summary>
//    /// Resets buffer to the set value - used for padding with 0s
//    /// </summary>
//    /// <param name="N"></param>
//    /// <param name="intBuffer"></param>
//    /// <param name="value"></param>
//    /// <returns></returns>
//    __global__ void kernResetIntBuffer(int N, int* intBuffer, int value) {
//        int index = (blockIdx.x * blockDim.x) + threadIdx.x;
//        if (index < N && index > -1) {
//            intBuffer[index] = value;
//        }
//    }
//
//    int blockSize = 32;
//
//    __global__ void kernUpSweep(int n, int* data, int d)
//    {
//        int idx = blockIdx.x * blockDim.x + threadIdx.x;
//        idx = idx * d - 1;
//        if (idx > n || idx < 0) return;
//        data[idx] += data[idx - (d >> 1)];
//    }
//
//    __global__ void kernChangeOneVal(int index, int* data, int val)
//    {
//        data[index] = val;
//    }
//
//    __global__ void kernDownSweep(int n, int* data, int d)
//    {
//        int idx = blockIdx.x * blockDim.x + threadIdx.x;
//        idx = idx * d - 1;
//        if (idx > n || idx < 0) return;
//        // Left child will become copy of parent
//        // Right child will be sum of left and parent
//        int left = idx - (d >> 1);
//        int t = data[left];
//        data[left] = data[idx];
//        data[idx] += t;
//    }
//
//    /**
//    * Performs stream compaction on idata, storing the result into odata.
//    * All zeroes are discarded.
//    *
//    * @param n      The number of elements in idata.
//    * @param odata  The array into which to store elements.
//    * @param idata  The array of elements to compact.
//    * @returns      The number of elements remaining after compaction.
//    */
//    int compact(int n, PathSegment* odata, const PathSegment* idata, std::function<bool(const PathSegment&)> predicate) {
//        int reqSize = ilog2ceil(n);
//        int ceil = 1 << reqSize;
//
//        dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
//        dim3 scanBlocksPerGrid((ceil + blockSize - 1) / blockSize);
//
//        PathSegment* dev_iArray;            // input array buffer
//        int* dev_boolArray;         // boolean array buffer
//        int* dev_boolScan;           // scan of the boolean array buffer
//        int* temp_Array;
//        PathSegment* dev_outArray;          // the output array buffer
//        hipMalloc((void**)&dev_boolArray, sizeof(int) * n);
//        hipMalloc((void**)&dev_iArray, sizeof(PathSegment) * n);
//        hipMalloc((void**)&dev_boolScan, sizeof(int) * ceil);
//        hipMalloc((void**)&dev_outArray, sizeof(PathSegment) * n);
//
//        hipMemcpy(dev_iArray, idata, sizeof(PathSegment) * n, hipMemcpyHostToDevice);
//
//        // Populate bool array
//        kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_boolArray, dev_iArray, predicate);
//
//        // Scan on bool array
//        hipMemcpy(dev_boolScan, dev_boolArray, sizeof(int) * n, hipMemcpyDeviceToDevice);
//        for (int d = 1; d < reqSize + 1; ++d)
//        {
//            scanBlocksPerGrid = dim3(((ceil >> (d - 1)) + blockSize - 1) / blockSize);
//            kernUpSweep << <scanBlocksPerGrid, blockSize >> > (ceil - 1, dev_boolScan, 1 << d);
//        }
//        kernChangeOneVal << <1, 1 >> > (ceil - 1, dev_boolScan, 0);
//        for (int d = reqSize; d > 0; --d)
//        {
//            scanBlocksPerGrid = dim3(((ceil >> (d - 1)) + blockSize - 1) / blockSize);
//            kernDownSweep << <scanBlocksPerGrid, blockSize >> > (ceil - 1, dev_boolScan, 1 << d);
//        }
//
//        // the resulting scan value was at a different location depending on padding or not, so the extra arithmetic here adjusts for that
//        hipMemcpy(temp_Array, dev_boolScan, sizeof(int) * (n + (n % 2)), hipMemcpyDeviceToHost);
//        int size = temp_Array[n - ((n + 1) % 2)];
//
//        // Compact
//        kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_outArray, dev_iArray, dev_boolArray, dev_boolScan);
//
//        hipMemcpy(odata, dev_outArray, sizeof(PathSegment) * n, hipMemcpyDeviceToHost);
//
//        hipFree(dev_boolArray);
//        hipFree(dev_iArray);
//        hipFree(dev_boolScan);
//        hipFree(dev_outArray);
//        return size;
//    }
//}