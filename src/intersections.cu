#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));

    return glm::length(r.origin - intersectionPoint);
}

// The body of this function is adapted from this paper:
// https://cadxfem.org/inf/Fast%20MinimumStorage%20RayTriangle%20Intersection.pdf
// backfaces are not culled atm
__host__ __device__ float triangleIntersectionTest(
    Triangle triangle,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    glm::vec2& uv,
    bool& outside)
{
    glm::vec3 edge1 = triangle.v1 - triangle.v0;
    glm::vec3 edge2 = triangle.v2 - triangle.v0;

    glm::vec3 pvec = glm::cross(r.direction, edge2);
    float det = glm::dot(edge1, pvec);

    if (det > -EPSILON && det < EPSILON) return -1;
    
    float inv_det = 1.f / det;
    glm::vec3 tvec = r.origin - triangle.v0;
    float u = glm::dot(tvec, pvec) * inv_det;
    if (u < 0.0f || u > 1.0f)
    {
        return -1;
    }

    glm::vec3 qvec = glm::cross(tvec, edge1);

    float v = glm::dot(r.direction, qvec) * inv_det;
    if (v < 0.0 || u + v > 1.0)
    {
        return -1;
    }

    float t = glm::dot(edge2, qvec) * inv_det;
    intersectionPoint = getPointOnRay(r, t);
    normal = glm::cross(edge1, edge2);
    outside = glm::dot(normal, r.direction) < EPSILON;

    float w = 1.0f - u - v;  // barycentric coordinate for vertex 0
    uv = w * triangle.uv0 + u * triangle.uv1 + v * triangle.uv2;
    uv = glm::fract(uv);    // enforced wrapping
    return t;
}
