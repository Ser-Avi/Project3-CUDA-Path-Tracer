#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));

    return glm::length(r.origin - intersectionPoint);
}

// The body of this function is adapted from this paper:
// https://cadxfem.org/inf/Fast%20MinimumStorage%20RayTriangle%20Intersection.pdf
// backfaces are not culled atm
__host__ __device__ float triangleIntersectionTest(
    Triangle triangle,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    glm::vec2& uv,
    bool& outside)
{
    glm::vec3 edge1 = triangle.v1 - triangle.v0;
    glm::vec3 edge2 = triangle.v2 - triangle.v0;

    glm::vec3 pvec = glm::cross(r.direction, edge2);
    float det = glm::dot(edge1, pvec);

    if (det > -EPSILON && det < EPSILON) return -1;
    
    float inv_det = 1.f / det;
    glm::vec3 tvec = r.origin - triangle.v0;
    float u = glm::dot(tvec, pvec) * inv_det;
    if (u < 0.0f || u > 1.0f)
    {
        return -1;
    }

    glm::vec3 qvec = glm::cross(tvec, edge1);

    float v = glm::dot(r.direction, qvec) * inv_det;
    if (v < 0.0 || u + v > 1.0)
    {
        return -1;
    }

    float t = glm::dot(edge2, qvec) * inv_det;
    intersectionPoint = getPointOnRay(r, t);
    normal = glm::cross(edge1, edge2);
    outside = glm::dot(normal, r.direction) < EPSILON;

    float w = 1.0f - u - v;  // barycentric coordinate for vertex 0
    uv = w * triangle.uv0 + u * triangle.uv1 + v * triangle.uv2;
    uv = glm::fract(uv);    // enforced wrapping
    return t;
}

__device__ bool IntersectAABB(const Ray& ray, const glm::vec3 bmin, const glm::vec3 bmax, float temp_T)
{
    float tx1 = (bmin.x - ray.origin.x) / ray.direction.x, tx2 = (bmax.x - ray.origin.x) / ray.direction.x;
    float tmin = min(tx1, tx2), tmax = max(tx1, tx2);
    float ty1 = (bmin.y - ray.origin.y) / ray.direction.y, ty2 = (bmax.y - ray.origin.y) / ray.direction.y;
    tmin = max(tmin, min(ty1, ty2)), tmax = min(tmax, max(ty1, ty2));
    float tz1 = (bmin.z - ray.origin.z) / ray.direction.z, tz2 = (bmax.z - ray.origin.z) / ray.direction.z;
    tmin = max(tmin, min(tz1, tz2)), tmax = min(tmax, max(tz1, tz2));
    return tmax >= tmin && tmin < temp_T && tmax > 0;
}

// In order for this function to be non-recursive, I used Sebastial Lague's method, as seen in this
// youtube video: https://www.youtube.com/watch?v=C1H4zIiCOaI&t=932s
__device__ float IntersectBVH(Ray& ray, const uint32_t nodeIdx, BVHNode* bvhNode, int* triIdx, Triangle* tri, float temp_t,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    glm::vec2& uv,
    bool& outside, int& idx)
{
    BVHNode nodeStack[16]; // max recursion depth is 16 for now.
    int stackIdx = 0;
    nodeStack[stackIdx++] = bvhNode[0];

    glm::vec3 tmp_p, tmp_nor;
    glm::vec2 tmp_uv;
    float min_t = FLT_MAX;

    while (stackIdx > 0)
    {
        BVHNode node = nodeStack[--stackIdx];

        if (IntersectAABB(ray, node.aabbMin, node.aabbMax, temp_t))
        {
            if (node.triCount > 0)  // i.e. is leaf
            {
                for (uint32_t i = node.leftFirst; i < node.leftFirst + node.triCount; ++i)
                {
                    temp_t = triangleIntersectionTest(tri[triIdx[node.leftFirst + i]], ray, tmp_p, tmp_nor, tmp_uv, outside);
                    if (temp_t < min_t && temp_t > 0.f)
                    {
                        intersectionPoint = tmp_p;
                        normal = tmp_nor;
                        uv = tmp_uv;
                        min_t = temp_t;
                        idx = i;
                    }
                }
            }
            else
            {
                nodeStack[stackIdx++] = bvhNode[node.leftFirst + 1];
                nodeStack[stackIdx++] = bvhNode[node.leftFirst + 0];
            }
        }
    }

    return min_t;
}
