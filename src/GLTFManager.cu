#include "hip/hip_runtime.h"
#include "GLTFManager.h"


TextureLoader::TextureLoader() = default;

TextureLoader::~TextureLoader() {
    cleanup();
}

hipTextureObject_t TextureLoader::loadTexture(const std::string& filename) {
    // Check cache first
    auto it = texture_cache.find(filename);
    if (it != texture_cache.end()) {
        return it->second;
    }

    std::cout << "Loading texture: " << filename << std::endl;

    // Load PNG using stb_image
    int width, height, channels;
    unsigned char* data = stbi_load(filename.c_str(), &width, &height, &channels, 4); // Force 4 channels

    if (!data) {
        std::cerr << "Failed to load texture: " << filename << std::endl;
        return 0;
    }

    std::cout << "Loaded: " << width << "x" << height << ", channels: " << channels << std::endl;

    // Create CUDA texture
    hipTextureObject_t tex_obj = createTextureFromData(data, width, height, 4);

    // Free CPU data
    stbi_image_free(data);

    if (tex_obj != 0) {
        texture_cache[filename] = tex_obj;
        std::cout << "Created CUDA texture object: " << tex_obj << std::endl;
    }

    return tex_obj;
}

hipTextureObject_t TextureLoader::getTexture(const std::string& filename) {
    return loadTexture(filename); // Load if not cached
}

hipTextureObject_t TextureLoader::createTextureFromData(const unsigned char* data,
    int width, int height, int channels) {
    hipTextureObject_t tex_obj = 0;
    hipArray_t cu_array = nullptr;

    // Create channel description
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<uchar4>();

    // Create CUDA array
    hipError_t err = hipMallocArray(&cu_array, &channel_desc, width, height);
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate CUDA array: " << hipGetErrorString(err) << std::endl;
        return 0;
    }

    // Copy data to array
    size_t pitch = width * channels * sizeof(unsigned char);
    err = hipMemcpy2DToArray(cu_array, 0, 0, data, pitch,
        width * channels * sizeof(unsigned char), height,
        hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Failed to copy texture data: " << hipGetErrorString(err) << std::endl;
        hipFreeArray(cu_array);
        return 0;
    }

    // Create resource description
    hipResourceDesc res_desc = {};
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = cu_array;

    // Create texture description
    hipTextureDesc tex_desc = {};
    tex_desc.addressMode[0] = hipAddressModeWrap;
    tex_desc.addressMode[1] = hipAddressModeWrap;
    tex_desc.filterMode = hipFilterModeLinear;
    tex_desc.readMode = hipReadModeNormalizedFloat; // Convert to [0,1]
    tex_desc.normalizedCoords = 1;
    tex_desc.maxAnisotropy = 1;

    // Create texture object
    err = hipCreateTextureObject(&tex_obj, &res_desc, &tex_desc, nullptr);
    if (err != hipSuccess) {
        std::cerr << "Failed to create texture object: " << hipGetErrorString(err) << std::endl;
        hipFreeArray(cu_array);
        return 0;
    }

    texture_arrays.push_back(cu_array);
    return tex_obj;
}

void TextureLoader::cleanup() {
    for (auto& pair : texture_cache) {
        if (pair.second != 0) {
            hipDestroyTextureObject(pair.second);
        }
    }
    texture_cache.clear();

    for (auto& array : texture_arrays) {
        if (array != nullptr) {
            hipFreeArray(array);
        }
    }
    texture_arrays.clear();
}

GLTFLoader::GLTFLoader() = default;

GLTFLoader::~GLTFLoader() {
    clear();
}

bool GLTFLoader::load(const std::string& filename) {
    clear();

    tinygltf::Model model;
    tinygltf::TinyGLTF loader;
    std::string err, warn;

    // Try loading as binary first, then ASCII
    bool success = loader.LoadBinaryFromFile(&model, &err, &warn, filename) ||
        loader.LoadASCIIFromFile(&model, &err, &warn, filename);

    if (!warn.empty()) {
        std::cout << "GLTF Warning: " << warn << std::endl;
    }
    if (!err.empty()) {
        std::cerr << "GLTF Error: " << err << std::endl;
    }
    if (!success) {
        std::cerr << "Failed to load GLTF file: " << filename << std::endl;
        return false;
    }

    if (!processModel(filename, model)) {
        std::cerr << "Failed to process GLTF model" << std::endl;
        return false;
    }

    std::cout << "Loaded GLTF: " << meshes.size() << " meshes, "
        << materials.size() << " materials, " << std::endl;

    return true;
}

void GLTFLoader::clear() {

    meshes.clear();
    materials.clear();
}

bool GLTFLoader::processModel(const std::string& filename, const tinygltf::Model& model) {
    // Extract directory for relative texture paths
    std::string dir = filename.substr(0, filename.find_last_of("/\\")) + "/";
    // Process materials first
    for (const auto& mat : model.materials) {
        materials.push_back(processMaterial(mat, model, dir));
    }

    // Add default material if none exist
    if (materials.empty()) {
        materials.push_back(MaterialData());
    }

    // Process meshes
    for (const auto& mesh : model.meshes) {
        for (const auto& primitive : mesh.primitives) {
            if (primitive.mode == TINYGLTF_MODE_TRIANGLES) {
                meshes.push_back(processPrimitive(primitive, model));
            }
        }
    }

    // Print texture paths for debugging
    for (size_t i = 0; i < materials.size(); i++) {
        if (!materials[i].base_color_texture_path.empty()) {
            std::cout << "Material " << i << " base color texture: "
                << materials[i].base_color_texture_path << std::endl;
        }
    }

    return true;
}

GLTFLoader::MaterialData GLTFLoader::processMaterial(const tinygltf::Material& mat,
    const tinygltf::Model& model, const std::string& dir) {
    MaterialData material;

    // Base color factor
    if (mat.pbrMetallicRoughness.baseColorFactor.size() == 4) {
        for (int i = 0; i < 4; i++) {
            material.base_color[i] = static_cast<float>(mat.pbrMetallicRoughness.baseColorFactor[i]);
        }
    }

    // Metallic/roughness factors
    material.metallic = static_cast<float>(mat.pbrMetallicRoughness.metallicFactor);
    material.roughness = static_cast<float>(mat.pbrMetallicRoughness.roughnessFactor);

    if (mat.pbrMetallicRoughness.baseColorTexture.index >= 0) {
        int tex_index = mat.pbrMetallicRoughness.baseColorTexture.index;
        if (tex_index < model.textures.size()) {
            const auto& texture = model.textures[tex_index];
            if (texture.source >= 0 && texture.source < model.images.size()) {
                const auto& image = model.images[texture.source];
                if (!image.uri.empty() && image.uri.find("data:") == std::string::npos) {
                    material.base_color_texture_path = dir + image.uri;
                }
            }
        }
    }

    return material;
}

GLTFLoader::MeshData GLTFLoader::processPrimitive(const tinygltf::Primitive& primitive,
    const tinygltf::Model& model) {
    MeshData mesh;
    mesh.material_id = primitive.material >= 0 ? primitive.material : 0;

    // Extract vertex attributes
    extractAttribute(primitive, model, "POSITION", mesh.vertices, 3);
    extractAttribute(primitive, model, "NORMAL", mesh.normals, 3);
    extractAttribute(primitive, model, "TEXCOORD_0", mesh.texcoords, 2);

    // Extract indices
    if (primitive.indices >= 0) {
        const auto& accessor = model.accessors[primitive.indices];
        const auto& bufferView = model.bufferViews[accessor.bufferView];
        const auto& buffer = model.buffers[bufferView.buffer];

        const unsigned char* data = buffer.data.data() + bufferView.byteOffset + accessor.byteOffset;

        if (accessor.componentType == TINYGLTF_COMPONENT_TYPE_UNSIGNED_INT) {
            const uint32_t* indices = reinterpret_cast<const uint32_t*>(data);
            mesh.indices.assign(indices, indices + accessor.count);
        }
        else if (accessor.componentType == TINYGLTF_COMPONENT_TYPE_UNSIGNED_SHORT) {
            const uint16_t* indices = reinterpret_cast<const uint16_t*>(data);
            for (size_t i = 0; i < accessor.count; i++) {
                mesh.indices.push_back(static_cast<uint32_t>(indices[i]));
            }
        }
        else if (accessor.componentType == TINYGLTF_COMPONENT_TYPE_UNSIGNED_BYTE) {
            const uint8_t* indices = reinterpret_cast<const uint8_t*>(data);
            for (size_t i = 0; i < accessor.count; i++) {
                mesh.indices.push_back(static_cast<uint32_t>(indices[i]));
            }
        }
    }

    return mesh;
}

void GLTFLoader::extractAttribute(const tinygltf::Primitive& primitive,
    const tinygltf::Model& model,
    const std::string& attribute,
    std::vector<float>& output, int components) {
    auto it = primitive.attributes.find(attribute);
    if (it == primitive.attributes.end()) return;

    int accessor_index = it->second;
    const auto& accessor = model.accessors[accessor_index];
    const auto& bufferView = model.bufferViews[accessor.bufferView];
    const auto& buffer = model.buffers[bufferView.buffer];

    const float* data = reinterpret_cast<const float*>(
        buffer.data.data() + bufferView.byteOffset + accessor.byteOffset);

    output.assign(data, data + accessor.count * components);
}

GLTFManager::GLTFManager() = default;

GLTFManager::~GLTFManager() {
    cleanup();
}

bool GLTFManager::uploadToGPU(const GLTFLoader& loader, TextureLoader& text_loader) {
    cleanup();

    uploadTriangles(loader.getMeshes());
    uploadMaterials(loader.getMaterials(), text_loader);

    std::cout << "Uploaded to GPU: " << num_triangles << " triangles, "
        << num_PBRmaterials << " materials" << std::endl;

    // delete loader as we will no longer need it
    loader.~GLTFLoader();
    return true;
}

void GLTFManager::cleanup() {
    if (dev_triangles) {
        hipFree(dev_triangles);
        dev_triangles = nullptr;
    }
    if (dev_PBRmaterials) {
        hipFree(dev_PBRmaterials);
        dev_PBRmaterials = nullptr;
    }

    num_triangles = 0;
    num_PBRmaterials = 0;
}

void GLTFManager::uploadTriangles(const std::vector<GLTFLoader::MeshData>& meshes) {
    std::vector<Triangle> host_triangles;

    for (const auto& mesh : meshes) {
        if (mesh.indices.size() % 3 != 0) continue;

        for (size_t i = 0; i < mesh.indices.size(); i += 3) {
            Triangle tri;
            uint32_t idx0 = mesh.indices[i];
            uint32_t idx1 = mesh.indices[i + 1];
            uint32_t idx2 = mesh.indices[i + 2];

            // Vertices
            tri.v0 = glm::vec3(mesh.vertices[idx0 * 3], mesh.vertices[idx0 * 3 + 1], mesh.vertices[idx0 * 3 + 2]);
            tri.v1 = glm::vec3(mesh.vertices[idx1 * 3], mesh.vertices[idx1 * 3 + 1], mesh.vertices[idx1 * 3 + 2]);
            tri.v2 = glm::vec3(mesh.vertices[idx2 * 3], mesh.vertices[idx2 * 3 + 1], mesh.vertices[idx2 * 3 + 2]);

            // Normals
            if (!mesh.normals.empty()) {
                tri.n0 = glm::vec3(mesh.normals[idx0 * 3], mesh.normals[idx0 * 3 + 1], mesh.normals[idx0 * 3 + 2]);
                tri.n1 = glm::vec3(mesh.normals[idx1 * 3], mesh.normals[idx1 * 3 + 1], mesh.normals[idx1 * 3 + 2]);
                tri.n2 = glm::vec3(mesh.normals[idx2 * 3], mesh.normals[idx2 * 3 + 1], mesh.normals[idx2 * 3 + 2]);
            }
            else {
                // Compute face normal if no normals provided
                glm::vec3 edge1 = glm::vec3(tri.v1.x - tri.v0.x, tri.v1.y - tri.v0.y, tri.v1.z - tri.v0.z);
                glm::vec3 edge2 = glm::vec3(tri.v2.x - tri.v0.x, tri.v2.y - tri.v0.y, tri.v2.z - tri.v0.z);
                glm::vec3 normal = glm::normalize(glm::cross(edge1, edge2));
                tri.n0 = tri.n1 = tri.n2 = normal;
            }

            // UV coordinates
            if (!mesh.texcoords.empty()) {
                tri.uv0 = glm::vec2(mesh.texcoords[idx0 * 2], mesh.texcoords[idx0 * 2 + 1]);
                tri.uv1 = glm::vec2(mesh.texcoords[idx1 * 2], mesh.texcoords[idx1 * 2 + 1]);
                tri.uv2 = glm::vec2(mesh.texcoords[idx2 * 2], mesh.texcoords[idx2 * 2 + 1]);
            }
            else {
                tri.uv0 = tri.uv1 = tri.uv2 = glm::vec2(0.0f, 0.0f);
            }

            tri.material_id = mesh.material_id;
            host_triangles.push_back(tri);
        }
    }

    num_triangles = host_triangles.size();
    if (num_triangles > 0) {
        hipMalloc(&dev_triangles, num_triangles * sizeof(Triangle));
        hipMemcpy(dev_triangles, host_triangles.data(),
            num_triangles * sizeof(Triangle), hipMemcpyHostToDevice);
    }
}

void GLTFManager::uploadMaterials(const std::vector<GLTFLoader::MaterialData>& materials,
    TextureLoader& text_loader) {
    std::vector<Material> host_materials;
    for (size_t i = 0; i < materials.size(); i++) {
        const auto& mat = materials[i];
        Material cuda_mat;

        cuda_mat.color = glm::vec3(mat.base_color[0], mat.base_color[1], mat.base_color[2]);
        cuda_mat.metallic = mat.metallic;
        cuda_mat.roughness = mat.roughness;

        // Load textures and get direct CUDA texture handles
        cuda_mat.base_color_tex = text_loader.getTexture(mat.base_color_texture_path);
        cuda_mat.metallic_roughness_tex = text_loader.getTexture(mat.metallic_roughness_texture_path);
        cuda_mat.normal_tex = text_loader.getTexture(mat.normal_texture_path);
        cuda_mat.emissive_tex = text_loader.getTexture(mat.emissive_texture_path);

        // Debug output
        std::cout << "Material " << i << ":" << std::endl;
        std::cout << "  Base color: (" << cuda_mat.color.x << ", "
            << cuda_mat.color.y << ", " << cuda_mat.color.z << ")" << std::endl;
        std::cout << "  Metallic: " << cuda_mat.metallic << ", Roughness: " << cuda_mat.roughness << std::endl;
        std::cout << "  Base color texture: " << (cuda_mat.base_color_tex != 0 ? "LOADED" : "MISSING")
            << " (handle: " << cuda_mat.base_color_tex << ")" << std::endl;
        if (!mat.base_color_texture_path.empty()) {
            std::cout << "  Texture path: " << mat.base_color_texture_path << std::endl;
        }

        host_materials.push_back(cuda_mat);
    }

    // Add a default material if no materials exist
    if (host_materials.empty()) {
        std::cout << "No materials found, creating default material" << std::endl;
        Material default_mat;
        default_mat.color = glm::vec3(0.8f, 0.8f, 0.8f);
        default_mat.metallic = 0.0f;
        default_mat.roughness = 0.5f;
        default_mat.emissive_factor = glm::vec3(0.0f);
        default_mat.base_color_tex = 0;  // No texture
        default_mat.metallic_roughness_tex = 0;
        default_mat.normal_tex = 0;
        default_mat.emissive_tex = 0;
        host_materials.push_back(default_mat);
    }

    num_PBRmaterials = host_materials.size();
    if (num_PBRmaterials > 0) {
        hipMalloc(&dev_PBRmaterials, num_PBRmaterials * sizeof(Material));
        hipMemcpy(dev_PBRmaterials, host_materials.data(),
            num_PBRmaterials * sizeof(Material), hipMemcpyHostToDevice);
        
    }
}